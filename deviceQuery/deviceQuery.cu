
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prints info about the device
// Takes in the device number and a pointer to the properties
void printProperties(int i, hipDeviceProp_t *prop){
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop->name );
}

int main( void ) {
	hipDeviceProp_t  *prop;
	int count;

	//Doesn't handle errors
	hipGetDeviceCount(&count);

	printf("Device Count: %d\n", count);

	//while(count > 0){
	for(int i = 0; i < count; i++){
		//Doesn't handle errors
		hipGetDeviceProperties( prop, i );
		printProperties(i, prop);	
		//count--;		
	}

	return 0;
}
