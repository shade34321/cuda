
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prints info about the device
// Takes in the device number and a pointer to the properties
void printProperties(int i, hipDeviceProp_t *prop){
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop->name );
}

int main(void){
	int count;
	hipDeviceProp_t  *prop;

	//Doesn't handle errors
	hipGetDeviceCount(&count);
	
	printf("Device Count: %d\n", count);

	while(count > 0){
		//Doesn't handle errors
		hipGetDeviceProperties( prop, count );
		printProperties(count, prop);	
		count--;		
	}

	return 0;
}
